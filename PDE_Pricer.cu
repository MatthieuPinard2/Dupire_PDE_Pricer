#include <iostream> 
#include <exception>

#include <hip/hip_runtime.h>
#include <string>

// Threads per Block, 256 is shown to give optimal results for the Kepler architecture.
#define TPB 256 

// Used for SOR/PSOR Kernel, 6 iterations have shown to provide with good accuracy at moderate cost with Omega = 1.3.
#define PSOR_ITERATIONS 6
#define PSOR_OMEGA 1.3f

// Bench CUDA Kernels using a templated C++ function
template <class fn>
void Bench(int Repetitions,       // The number of times the benchmark has to be launched.
           std::string Message,   // A message to be appended to the output (can be the name of the module, or the params)
           fn&& Function) {       // The function.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_elapsed_time = 0.f;
    // Sequentially invoke the function.
    for (int i = 0; i < Repetitions; ++i) {
        hipEventRecord(start);
        Function();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float iteration_elapsed_time = 0.f;
        hipEventElapsedTime(&iteration_elapsed_time, start, stop);
        total_elapsed_time += iteration_elapsed_time;
    }
    // Display the average of running times
    std::cout << Message << ": Time Elapsed = " << total_elapsed_time / double(Repetitions) << " ms \n";
}

// Wrapper around cudaMalloc().
inline float * cudaAlloc(size_t n) {
    float* Ptr = 0;
    hipMalloc(&Ptr, n * sizeof(float));
    if (!Ptr)
        throw std::bad_alloc();
    return Ptr;
}

// Kernel to initialize Saxis (Saxis[i] = Smin + i * deltaS)
__global__ void Saxis_Initialize(float * Saxis,
                                 const float Smin,
                                 const float deltaS,
                                 const int Length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < Length) {
        Saxis[i] = Smin + i * deltaS;
    }
}

// Computes P[i] = max(0, S[i] - K) for i in 0:(SizeS-1)
__global__ void Call(float * __restrict__ P,        // The payoff (output)
                     const float * __restrict__ S,  // The underlying level (input)
                     const float K,                 // The option Strike (input)
                     const int SizeS) {             // Size of the S-axis (input)
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeS) {
        P[i] = max(S[i] - K, 0.f);
    }
}

// Refer to Call.
__global__ void Put(float * __restrict__ P,
                    const float * __restrict__ S,
                    const float K,
                    const int SizeS) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeS) {
        P[i] = max(K - S[i], 0.f);
    }
}

// Computes the Dirichlet boundary of an European Call for the upper-bound of the S-axis, for t in 0:(SizeT-1).
__global__ void MaxBoundary_EuropeanCall(float * __restrict__ P,        // The Dirichlet Boundary (output)
                                         const float maxS,              // Maximum of S-axis (input, should be K + 6 * Vol * sqrt(T))
                                         const float * __restrict__ r,  // The interest rate (input)
                                         const float * __restrict__ q,  // The dividend yield (input)
                                         const float K,                 // The option strike (input)
                                         const int SizeT) {             // Size of the T-axis (input) 
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeT) {
        P[i] = maxS *__expf(q[i]) - K * __expf(r[i]);
    }
}

// Refer to MaxBoundary_EuropeanCall.
__global__ void MaxBoundary_AmericanCall(float * __restrict__ P,
                                         const float maxS,
                                         const float K,
                                         const int SizeT) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeT) {
        P[i] = maxS - K;
    }
}

// Refer to MaxBoundary_EuropeanCall.
__global__ void MinBoundary_EuropeanPut(float * __restrict__ P,
                                        const float minS,
                                        const float * __restrict__ r,
                                        const float * __restrict__ q,
                                        const float K,
                                        const int SizeT) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeT) {
        P[i] = -minS * __expf(q[i]) + K * __expf(r[i]);
    }
}

// Refer to MaxBoundary_EuropeanCall.
__global__ void MinBoundary_AmericanPut(float * __restrict__ P,
                                        const float minS,
                                        const float K,
                                        const int SizeT) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < SizeT) {
        P[i] = -minS + K;
    }
}

// Handler for Explicit kernels. It returns the value of the derivative at T-1 given the value of the derivative at T and 
// local parameters.
__inline__ __device__ float ExplicitHandler(const float * __restrict__ pastV,   // Value of the derivative at T 
                                            const float S,                      // The underlying level
                                            const float r,                      // Instantaneous interest rate
                                            const float q,                      // Instantanous dividend yield 
                                            const float sigma,                  // Local volatility
                                            const float deltaT,                 // Time step
                                            const float deltaS,                 // Space step
                                            const float minBoundary,            // Min Dirichlet boundary at T
                                            const float maxBoundary,            // Max Dirichlet boundary at T
                                            const int i,                        // Space index
                                            const int SizeS) {                  // Size of S-axis
    // Computation of the BS-Dupire coefficients
    float Base = (deltaT * S) / (2.f * deltaS);
    float Order1 = Base * (r - q);
    float Order2 = (Base * sigma * sigma * S) / deltaS;
    float Alpha = Order1 - Order2;
    float Beta = 1.f + r * deltaT + 2.f * Order2;
    float Gamma = -Order1 - Order2;
    float Mid = pastV[i];
    float Down, Up;
    if (i == 0) {
        // Lower bound.
        Down = minBoundary;
        Up = pastV[i + 1];
    }
    else if (i == SizeS - 1) {
        // Upper bound.
        Down = pastV[i - 1];
        Up = maxBoundary;
    }
    else {
        // Tridiagonal case.
        Down = pastV[i - 1];
        Up = pastV[i + 1];
    }
    return (Alpha * Down + Beta * Mid + Gamma * Up);
}

// Kernel for explicit scheme applied to American options.
__global__ void ExplicitKernel_American(float * __restrict__ Grid,		// The PDE grid (output)
                                        const float * __restrict__ S,		// Value of asset
                                        const float * __restrict__ r,		// Risk-free rate
                                        const float * __restrict__ q,		// Dividend yield
                                        const float * __restrict__ sigma,		// Volatility
                                        const float deltaT,			// Time precision
                                        const float deltaS,			// Asset precision
                                        const float * __restrict__ minBoundary,         // Values of derivative for S = minS
                                        const float * __restrict__ maxBoundary,         // Values of derivative for S = maxS
                                        const int sizeS, const int sizeT) {		// The number of elements for S-axis and T-axis
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < sizeS) {
        float* pastV = Grid;
        float* currV = pastV + sizeS;
        float* Payoff = Grid;
        // Iterate over time steps. j = 0 equals the maturity, and j = sizeT - 1 is today.
        for (int j = 1; j < sizeT; ++j) {
            float X = ExplicitHandler(pastV,
                                      S[i],
                                      r[j],
                                      q[j],
                                      sigma[i + j * sizeS],
                                      deltaT,
                                      deltaS,
                                      minBoundary[j],
                                      maxBoundary[j],
                                      i,
                                      sizeS);
            // Take the maximum value btw. Payoff and derivative value
            currV[i] = max(X, Payoff[i]);
            // Update currV, pastV and synchronize within block.
            pastV = currV;
            currV += sizeS;
            // Synchronize the block so all threads are in the same time step.
            __syncthreads();
        }
    }
}

// Kernel for explicit scheme applied to European options.
__global__ void ExplicitKernel_European(float * __restrict__ Grid,		// The PDE grid (output)
                                        const float * __restrict__ S,		// Value of asset
                                        const float * __restrict__ r,		// Risk-free rate
                                        const float * __restrict__ q,		// Dividend yield
                                        const float * __restrict__ sigma,		// Volatility
                                        const float deltaT,			// Time precision
                                        const float deltaS,			// Asset precision
                                        const float * __restrict__ minBoundary,         // Values of derivative for S = minS
                                        const float * __restrict__ maxBoundary,         // Values of derivative for S = maxS
                                        const int sizeS, const int sizeT) {		// The number of elements for S-axis and T-axis
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < sizeS) {
        float* pastV = Grid;
        float* currV = pastV + sizeS;
        // Iterate over time steps. j = 0 equals the maturity, and j = sizeT - 1 is today.
        for (int j = 1; j < sizeT; ++j) {
            currV[i] = ExplicitHandler(pastV,
                                       S[i],
                                       r[j],
                                       q[j],
                                       sigma[i + j * sizeS],
                                       deltaT,
                                       deltaS,
                                       minBoundary[j],
                                       maxBoundary[j],
                                       i,
                                       sizeS);
            // Update currV, pastV and synchronize within block.
            pastV = currV;
            currV += sizeS;
            // Synchronize the block so all threads are in the same time step.
            __syncthreads();
        }
    }
}

// Handler for Implicit kernels. Basically computes the tridiagonal system given the value of the derivative at T and 
// local parameters and returns new_val
__inline__ __device__ float ImplicitHandler(float * __restrict__ newV,     // Is set to new_val. (output)
                                            float * __restrict__ Array,    // Shared memory. Is set to new_val (output)
                                            float & __restrict__ _LD,      // The lower diagonal element (output)
                                            float & __restrict__ _D,       // The diagonal element (output)
                                            float & __restrict__ _UD,      // The upper diagonal element (output)
                                            const float sigma,             // Local volatility
                                            const float new_val,           // The value of derivative at T
                                            const float S,                 // Underlying level
                                            const float deltaT,            // Time step
                                            const float deltaS,            // Space step
                                            const float _r,                // Instantaneous interest rate
                                            const float _q,                // Instantaneous dividend yield
                                            const float minBoundary,       // Min Dirichlet boundary at T
                                            const float maxBoundary,       // Max Dirichlet boundary at T
                                            const int i,                   // Space index
                                            const int SizeS) {             // Size of S-axis
    // Computation of the tridiagnonal value given the BS-Dupire parameters
    newV[i] = new_val;
    float old_val = new_val;
    float Base = (deltaT * S) / (2.f * deltaS);
    float Order1 = Base * (_r - _q);
    float Order2 = (Base * sigma * sigma * S) / deltaS;
    _LD = Order2 - Order1;
    _D = 1.f - _r * deltaT - 2.f * Order2;
    _UD = Order2 + Order1;
    // Apply Dirichlet boundaries.
    if (i == 0) {
        old_val -= _LD * minBoundary;
    }
    else if (i == SizeS - 1) {
        old_val -= _UD * maxBoundary;
    }
    // Shared memory optimization...
    Array[i] = new_val;
    return old_val;
}

// Computes an iteration of the SOR algorithm. 
__inline__ __device__ float SORHandler(const float * __restrict__ Array,     // "Neighbours" previous iteration values
                                       const float old_val,                  // Previous iteration value
                                       const float _LD,                      // The lower diagonal element
                                       const float _D,                       // The diagonal element
                                       const float _UD,                      // The upper diagonal element
                                       const int i,                          // Space index
                                       const int SizeS) {                    // Size of S-axis
    float Up = (i > SizeS - 2) ? 0.f : Array[i + 1];
    float Down = (i < 1) ? 0.f : Array[i - 1];
    float r = (1.f - PSOR_OMEGA) * Array[i] + (PSOR_OMEGA / _D) * (old_val - _LD * Down - _UD * Up);
    return r;
}

// Kernel for implicit scheme applied to European options.
__global__ void ImplicitKernel_European(float * __restrict__ NewValues,                   // The PDE grid, starting at T-1 (it is offset by SizeS) (output)
                                        const float * __restrict__ R,                     // Instantaneous interest rate
                                        const float * __restrict__ Q,                     // Instantaneous dividend yield
                                        const float * __restrict__ _Sigma,                // Local Volatility
                                        const float * __restrict__ _S,                    // S-axis
                                        const float * __restrict__ minBoundary,           // Min Dirichlet boundary at each time step
                                        const float * __restrict__ maxBoundary,           // Max Dirichlet boundary at each time step
                                        const float deltaT,                               // Time step
                                        const float deltaS,                               // Space step
                                        const int SizeS, const int SizeT) {               // Sizes of S- and T-axes
    extern __shared__ float Array[];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float _D, _UD, _LD, old_val, S;
    if (i < SizeS) {
        S = _S[i];
        // Iterate over time steps. j = 0 equals the maturity, and j = sizeT - 1 is today.
        for (int j = 1; j < SizeT; ++j) {
            old_val = ImplicitHandler(NewValues, Array, _LD, _D, _UD,
                                      _Sigma[i], NewValues[i - SizeS], S,
                                      deltaT, deltaS, R[j], Q[j], minBoundary[j], 
                                      maxBoundary[j], i, SizeS);
            // This synchronization is used so all threads are in the same time step!
            __syncthreads();
            // Case "Red" : even threads
            if (!(i & 1)) {
                // Iterate PSOR_ITERATIONS times
                for (int k = 0; k < PSOR_ITERATIONS; ++k) {
                    Array[i] = SORHandler(Array, old_val, _LD, _D, _UD, i, SizeS);
                    // Synchronize Black and Red threads.
                    __syncthreads();
                    // While "Black" threads are computing, do nothing.
                    // Synchronize Black and Red threads.
                    __syncthreads();
                }
            }
            // Case "Black" : odd threads
            else {
                // Iterate PSOR_ITERATIONS times
                for (int k = 0; k < PSOR_ITERATIONS; ++k) {
                    // While "Red" threads are computing, do nothing.
                    // Synchronize Black and Red threads.
                    __syncthreads();
                    Array[i] = SORHandler(Array, old_val, _LD, _D, _UD, i, SizeS);
                    // Synchronize Black and Red threads.
                    __syncthreads();
                }
            }
            NewValues[i] = Array[i];
            NewValues += SizeS;
            _Sigma += SizeS;
        }
    }
}
        
// Kernel for implicit scheme applied to American options. Refer to ImplicitKernel_European for parameters and comments.
__global__ void ImplicitKernel_American(float * __restrict__ NewValues,
                                        const float * __restrict__ R,
                                        const float * __restrict__ Q,
                                        const float * __restrict__ _Sigma,
                                        const float * __restrict__ _S,
                                        const float * __restrict__ minBoundary,
                                        const float * __restrict__ maxBoundary,
                                        const float deltaT,
                                        const float deltaS,
                                        const int SizeS, const int SizeT) {
    extern __shared__ float Array[];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float r, _Payoff, _D, _UD, _LD;
    float old_val, S;
    if (i < SizeS) {
        _Payoff = NewValues[i - SizeS];
        S = _S[i];
        // Iterate for each time step using an implicit PSOR scheme.
        for (int j = 1; j < SizeT; ++j) {
            old_val = ImplicitHandler(NewValues, Array, _LD, _D, _UD,
                                      _Sigma[i], NewValues[i - SizeS], S,
                                      deltaT, deltaS, R[j], Q[j], minBoundary[j], 
                                      maxBoundary[j], i, SizeS);
            __syncthreads();
            if (!(i & 1)) {
                for (int k = 0; k < PSOR_ITERATIONS; ++k) {
                    r = SORHandler(Array, old_val, _LD, _D, _UD, i, SizeS);
                    // This maximum condition guarantees the American exercise-style of the option !
                    r = max(_Payoff, r);
                    Array[i] = r;
                    __syncthreads();
                    __syncthreads();
                }
            }
            else {
                for (int k = 0; k < PSOR_ITERATIONS; ++k) {
                    __syncthreads();
                    r = SORHandler(Array, old_val, _LD, _D, _UD, i, SizeS);
                    // This maximum condition guarantees the American exercise-style of the option !
                    r = max(_Payoff, r);
                    Array[i] = r;
                    __syncthreads();
                }
            }
            NewValues[i] = Array[i];
            NewValues += SizeS;
            _Sigma += SizeS;
        }
    }
}

class BlackScholesPDE {
private:
    int sizeS;		// Dimension of grid (S-axis)
    int sizeT;		// Dimension of grid (T-axis)
    float minT;		// Minimum value of T
    float maxT;		// Maximum value of T
    float minS;		// Minimum value of S
    float maxS;		// Maximum value of S
    float* sigma;		// Local Volatility. 
                                 // Should be given as a row-ordered matrix with 
                                 // Rows = sizeT (1st row <=> t = T and last row <=> t = 0), Cols = sizeS
    float* r;		// Instantaneous Risk-free rate
                                 // Should be given as a vector of size sizeT, with 1st element <=> t = T and last element <=> t = 0
    float* q;	           // Instantaneous Dividend yield
                                 // Should be given as a vector of size sizeT, with 1st element <=> t = T and last element <=> t = 0
    float deltaT;		// Implied from minT, maxT and sizeT
    float deltaS;		// Implied from minS, maxS and sizeS
    float* Grid;		// Stores the PDE grid into the GPU.
    float* cpuGrid;		// Stores the PDE grid into the CPU.
    float* MinBoundary;	// Stores the Dirichlet boundary for S = minS.
    float* MaxBoundary;	// Stores the Dirichlet boundary for S = maxS.
    float* Saxis;		// Stores the values taken by the asset.
    float* Integral_r;	// Stores the integrated risk-free rate.
    float* Integral_q;	// Stores the integrated dividend yield.
    float* Integral_r_CPU;	// Stores the integrated risk-free rate in central memory.
    float* Integral_q_CPU;	// Stores the integrated dividend yield in central memory.
public:
    BlackScholesPDE(
        int _sizeT, int _sizeS,
        float _minT, float _maxT,
        float _minS, float _maxS,
        float* _r, float* _q, float* _sigma);
    ~BlackScholesPDE();
    void CopyToCPU();
    void Print_S(int timeIdx);
    void Print_T(int assetIdx);
    void EuropeanCall(float K);
    void EuropeanPut(float K);
    void AmericanCall(float K);
    void AmericanPut(float K);
    void ImplicitSolving_European();
    void ImplicitSolving_American();
    void ExplicitSolving_European();
    void ExplicitSolving_American();
};

// CTOR.
BlackScholesPDE::BlackScholesPDE(int _sizeT, int _sizeS,
                                 float _minT, float _maxT,
                                 float _minS, float _maxS,
                                 float* _r, float* _q, float* _CPULocalVol) :
    sizeT(_sizeT), sizeS(_sizeS),
    minT(_minT), maxT(_maxT),
    minS(_minS), maxS(_maxS) {
    // deltaT is made negative as the t-axis goes backward.
    deltaT = (minT - maxT) / (sizeT - 1);
    deltaS = (maxS - minS) / (sizeS - 1);
    // Allocate the grid on both GPU and CPU.
    Grid = cudaAlloc(sizeT * sizeS);
    cpuGrid = (float*)malloc(sizeT * sizeS * sizeof(float));
    // Storage for Dirichlet boundaries.
    MinBoundary = cudaAlloc(sizeT);
    MaxBoundary = cudaAlloc(sizeT);
    // Allocate and initialize S-axis.
    Saxis = cudaAlloc(sizeS);
    Saxis_Initialize<<<(sizeS + TPB - 1) / TPB, TPB>>>(Saxis, minS, deltaS, sizeS);
    // Same for local volatility and rates
    sigma = cudaAlloc(sizeS * sizeT);
    hipMemcpy(sigma, _CPULocalVol, sizeS * sizeT * sizeof(float), hipMemcpyHostToDevice);
    r = cudaAlloc(sizeT);
    hipMemcpy(r, _r, sizeT * sizeof(float), hipMemcpyHostToDevice);
    q = cudaAlloc(sizeT);
    hipMemcpy(q, _q, sizeT * sizeof(float), hipMemcpyHostToDevice);
    // r, q are instantanous rates. We have to integrate them to have the "correct" ZC curve.
    Integral_r_CPU = (float*)malloc(sizeT * sizeof(float));
    Integral_q_CPU = (float*)malloc(sizeT * sizeof(float));
    Integral_r = cudaAlloc(sizeT);
    Integral_q = cudaAlloc(sizeT);
    Integral_r_CPU[0] = 0.f;
    Integral_q_CPU[0] = 0.f;
    for (int i = 1; i < sizeT; ++i) {
        Integral_r_CPU[i] = Integral_r_CPU[i - 1] - deltaT * 0.5f * (_r[i] + _r[i - 1]);
        Integral_q_CPU[i] = Integral_q_CPU[i - 1] - deltaT * 0.5f * (_q[i] + _q[i - 1]);
    }
    hipMemcpy(Integral_q, Integral_q_CPU, sizeT * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Integral_r, Integral_r_CPU, sizeT * sizeof(float), hipMemcpyHostToDevice);
}

// DTOR. Basically frees all memory.
BlackScholesPDE::~BlackScholesPDE() {
    hipFree(Grid);
    hipFree(MinBoundary);
    hipFree(MaxBoundary);
    hipFree(Saxis);
    hipFree(sigma);
    hipFree(q);
    hipFree(r);
    hipFree(Integral_r);
    hipFree(Integral_q);
    free(cpuGrid);
    free(Integral_r_CPU);
    free(Integral_q_CPU);
}

// Initializes the PDE Solver with an European Call of strike K.
void BlackScholesPDE::EuropeanCall(float K) {
    // Case i = 0 solved by payoff function
    Call<<<(sizeS + TPB - 1) / TPB, TPB>>>(Grid, Saxis, K, sizeS);
    // Dirichlet boundaries.
    // For European Call, S(t) = minS => Call(t) = 0.
    hipMemset(MinBoundary, 0, sizeT * sizeof(float));
    // For European Call, S(t) = maxS => Call(T) = maxS * exp(-qT) - K * exp(-rT)
    MaxBoundary_EuropeanCall<<<(sizeT + TPB - 1) / TPB, TPB>>>(MaxBoundary,
                                                               maxS,
                                                               Integral_r,
                                                               Integral_q,
                                                               K,
                                                               sizeT);
}

// Initializes the PDE Solver with an European Put of strike K.
void BlackScholesPDE::EuropeanPut(float K) {
    // Case i = 0 solved by payoff function
    Put<<<(sizeS + TPB - 1) / TPB, TPB>>>(Grid, Saxis, K, sizeS);
    // Dirichlet boundaries.
    // For European Put, S(t) = maxS => Put(t) = 0.
    hipMemset(MaxBoundary, 0, sizeT * sizeof(float));
    // For European Put, S(t) = minS => Put(T) = - minS * exp(-qT) + K * exp(-rT)
    MinBoundary_EuropeanPut<<<(sizeT + TPB - 1) / TPB, TPB>>>(MinBoundary,
                                                              minS,
                                                              Integral_r,
                                                              Integral_q,
                                                              K,
                                                              sizeT);
}

// Initializes the PDE Solver with an American Call of strike K.
void BlackScholesPDE::AmericanCall(float K) {
    // Case i = 0 solved by payoff function
    Call<<<(sizeS + TPB - 1) / TPB, TPB>>>(Grid, Saxis, K, sizeS);
    // Dirichlet boundaries.
    // For American Call, S(t) = minS => Call(t) = 0.
    hipMemset(MinBoundary, 0, sizeT * sizeof(float));
    // For American Call, S(t) = maxS => Call(t) = maxS - K
    MaxBoundary_AmericanCall<<<(sizeT + TPB - 1) / TPB, TPB>>>(MaxBoundary,
                                                               maxS,
                                                               K,
                                                               sizeT);
}

// Initializes the PDE Solver with an American Put of strike K.
void BlackScholesPDE::AmericanPut(float K) {
    // Case i = 0 solved by payoff function
    Put<<<(sizeS + TPB - 1) / TPB, TPB>>>(Grid, Saxis, K, sizeS);
    // Dirichlet boundaries.
    // For American Put, S(t) = maxS => Put(t) = 0.
    hipMemset(MaxBoundary, 0, sizeT * sizeof(float));
    // For American Put, S(t) = minS => Put(t) = - minS + K
    MinBoundary_AmericanPut<<<(sizeT + TPB - 1) / TPB, TPB>>>(MinBoundary,
                                                              minS,
                                                              K,
                                                              sizeT);
}

// Copy the grid back to central memory.
void BlackScholesPDE::CopyToCPU() {
    // Copy the grid back to Central memory
    hipMemcpy(cpuGrid, Grid, sizeS * sizeT * sizeof(float), hipMemcpyDeviceToHost);
}

// Prints the derivative value at a fixed time specified by timeIdx.
void BlackScholesPDE::Print_S(int timeIdx) {
    for (int i = 0; i < sizeS; ++i) {
        // The timeIdx goes backward, ie. timeIdx = 0 at maturity, and = (sizeT - 1) at option issue.
        std::cout << minS + i * deltaS << "\t" << cpuGrid[timeIdx * sizeS + i] << "\n";
    }
}

// Prints the derivative value at a fixed asset level specified by assetIdx.
void BlackScholesPDE::Print_T(int assetIdx) {
    for (int i = 0; i < sizeT; ++i) {
        std::cout << minT + i * deltaT << "\t" << cpuGrid[i * sizeS + assetIdx] << "\n";
    }
}

void BlackScholesPDE::ExplicitSolving_European() {
    // 1 block is a limitation to synchronizing all the threads at each time step.
    ExplicitKernel_European<<<1, sizeS>>>(Grid,
                                          Saxis,
                                          r, q, sigma, deltaT, deltaS,
                                          MinBoundary, MaxBoundary, sizeS, sizeT);
}

void BlackScholesPDE::ExplicitSolving_American() {
    // 1 block is a limitation to synchronizing all the threads at each time step.
    ExplicitKernel_American<<<1, sizeS>>>(Grid,
                                          Saxis,
                                          r, q, sigma, deltaT, deltaS,
                                          MinBoundary, MaxBoundary, sizeS, sizeT);
}

void BlackScholesPDE::ImplicitSolving_European() {
    // 1 block is a limitation to synchronizing all the threads at each time step.
    ImplicitKernel_European<<<1, sizeS, sizeS * sizeof(float)>>>(Grid + sizeS, 
                                                                 r, q, sigma, Saxis,
                                                                 MinBoundary, MaxBoundary, 
                                                                 deltaT, deltaS, sizeS, sizeT);
}

void BlackScholesPDE::ImplicitSolving_American() {
    // 1 block is a limitation to synchronizing all the threads at each time step.
    ImplicitKernel_American<<<1, sizeS, sizeS * sizeof(float)>>>(Grid + sizeS, 
                                                                 r, q, sigma, Saxis,
                                                                 MinBoundary, MaxBoundary, 
                                                                 deltaT, deltaS, sizeS, sizeT);
}

// Main function.
int main() {
    // PDE Parameters.
    int sizeT = 20000;
    int sizeS = 5 * 190 + 1;
    float minT = 0.0f;
    float maxT = 1.0f;
    // Should be in the range [max(0, K-6*Sigma*sqrt(T)); K+6*Sigma*sqrt(T)]
    float minS = 0.0f;
    float maxS = 190.0f;

    // Market observed parameters.
    float* sigma = (float*)malloc(sizeT * sizeS * sizeof(float));
    float* r = (float*)malloc(sizeT * sizeof(float));
    float* q = (float*)malloc(sizeT * sizeof(float));
    for (int i = 0; i < sizeT; ++i) {
        r[i] = 0.064f;
        q[i] = 0.045f;
    }
    for (int i = 0; i < sizeT * sizeS; ++i) {
        sigma[i] = 0.15f;
    }

    // Solving part.
    BlackScholesPDE Solver(
        sizeT, sizeS,
        minT, maxT,
        minS, maxS,
        r, q, sigma);
    float Strike = 100.f;
    Solver.EuropeanPut(Strike);
    //Solver.AmericanPut(Strike);
    Bench(50, "ImplicitSolving_European", [&]() {
        //Solver.ExplicitSolving_European();
        //Solver.ExplicitSolving_American();
        Solver.ImplicitSolving_European();
        //Solver.ImplicitSolving_American();
        hipDeviceSynchronize();
    });
    Solver.CopyToCPU();
    
    // Print at t = 0.
    Solver.Print_S(sizeT - 1);
    getchar();
    free(sigma);
    free(r);
    free(q);
    return 0;
}
